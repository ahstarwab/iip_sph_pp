#include "hip/hip_runtime.h"
/*
 * ===========================================================
 *           Copyright (c) 2018, __IIPLAB__
 *                All rights reserved.
 * 
 * This Source Code Form is subject to the terms of
 * the Mozilla Public License, v. 2.0. 
 * If a copy of the MPL was not distributed with this file,
 *  You can obtain one at http://mozilla.org/MPL/2.0/.
 * ===========================================================
 */
//#include "mother.h"
#include "iip_matrix.h"
#include "hip/hip_runtime.h"
#include ""

#if DEBUG
#define CUDA_CALL(x) \
{ \
	const hipError_t a = (x); \
	if(a != hipSuccess) { \
		printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); \
		hipDeviceReset(); assert(0); \
	} \
}
#else
#define CUDA_CALL(x) {(x);}
#endif
hipblasHandle_t handle;
UINT max_thread;
UINT max_block;


/**** allocMAT  ****/
MAT* alloc_MAT_1d(UINT d0)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	MAT* mat = (MAT*)malloc(sizeof(MAT));
	mat->ndim = 0;
	mat->d0 = d0;
	mat->d1 = 1;
	mat->d2 = 1;
	hipMalloc((void**)&(mat->data), sizeof(DTYPE) * d0);

	return mat;
}
MAT* alloc_MAT_2d(UINT d0, UINT d1)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	MAT* mat = (MAT*)malloc(sizeof(MAT));
	mat->ndim = 1;
	mat->d0 = d0;
	mat->d1 = d1;
	mat->d2 = 1;
	hipMalloc((void**)&(mat->data), sizeof(DTYPE) * d0* d1);

	return mat;

}
MAT* alloc_MAT_3d(UINT d0, UINT d1, UINT d2)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	MAT* mat = (MAT*)malloc(sizeof(MAT));
	mat->ndim = 2;
	mat->d0 = d0;
	mat->d1 = d1;
	mat->d2 = d2;
	hipMalloc((void**)&(mat->data), sizeof(DTYPE) * d0* d1*d2);

	return mat;
}

CMAT* alloc_CMAT_1d(UINT d0)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	CMAT* mat = (CMAT*)malloc(sizeof(CMAT));
	mat->ndim = 0;
	mat->d0 = d0;
	mat->d1 = 1;
	mat->d2 = 1;
	hipMalloc((void**)&(mat->data), sizeof(CTYPE) * d0);

	return mat;
}
CMAT* alloc_CMAT_2d(UINT d0, UINT d1)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	CMAT* mat = (CMAT*)malloc(sizeof(CMAT));
	mat->ndim = 1;
	mat->d0 = d0;
	mat->d1 = d1;
	mat->d2 = 1;
	hipMalloc((void**)&(mat->data), sizeof(CTYPE) * d0* d1);

	return mat;
}
CMAT* alloc_CMAT_3d(UINT d0, UINT d1, UINT d2)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	CMAT* mat = (CMAT*)malloc(sizeof(CMAT));
	mat->ndim = 2;
	mat->d0 = d0;
	mat->d1 = d1;
	mat->d2 = d2;
	hipMalloc((void**)&(mat->data), sizeof(CTYPE) * d0* d1* d2);

	return mat;
}
/**** zeros  ****/

MAT* zeros_1d(UINT d0)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	MAT* mat = (MAT*)malloc(sizeof(MAT));
	mat->ndim = 0;
	mat->d0 = d0;
	mat->d1 = 1;
	mat->d2 = 1;
	hipMalloc((void**)&(mat->data), sizeof(DTYPE) * d0);
	hipMemset(mat->data, 0, sizeof(DTYPE)*d0);
	return mat;
}

MAT* zeros_2d(UINT d0, UINT d1)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	MAT* mat = (MAT*)malloc(sizeof(MAT));
	mat->ndim = 1;
	mat->d0 = d0;
	mat->d1 = d1;
	mat->d2 = 1;

	hipMalloc((void**)&(mat->data), sizeof(DTYPE) * d0 *d1);
	hipMemset(mat->data, 0, sizeof(DTYPE)*d0*d1);

	return mat;
}
MAT* zeros_3d(UINT d0, UINT d1, UINT d2)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	MAT* mat = (MAT*)malloc(sizeof(MAT));
	mat->ndim = 2;
	mat->d0 = d0;
	mat->d1 = d1;
	mat->d2 = d2;

	hipMalloc((void**)&(mat->data), sizeof(DTYPE) * d0 *d1 *d2);
	hipMemset(mat->data, 0, sizeof(DTYPE)*d0*d1*d2);
	return mat;
}


CMAT* czeros_1d(UINT d0)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	CMAT* mat = (CMAT*)malloc(sizeof(CMAT));
	mat->ndim = 0;
	mat->d0 = d0;
	mat->d1 = 1;
	mat->d2 = 1;

	hipMalloc((void**)&(mat->data), sizeof(CTYPE) * d0);
	hipMemset(mat->data, 0, sizeof(CTYPE)*d0);
	return mat;
}

CMAT* czeros_2d(UINT d0, UINT d1)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	CMAT* mat = (CMAT*)malloc(sizeof(CMAT));
	mat->ndim = 1;
	mat->d0 = d0;
	mat->d1 = d1;
	mat->d2 = 1;

	hipMalloc((void**)&(mat->data), sizeof(CTYPE) * d0 *d1);
	hipMemset(mat->data, 0, sizeof(CTYPE)*d0*d1);
	return mat;
}
CMAT* czeros_3d(UINT d0, UINT d1, UINT d2)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	CMAT* mat = (CMAT*)malloc(sizeof(CMAT));
	mat->ndim = 2;
	mat->d0 = d0;
	mat->d1 = d1;
	mat->d2 = d2;

	hipMalloc((void**)&(mat->data), sizeof(CTYPE) * d0 *d1*d2);
	hipMemset(mat->data, 0, sizeof(CTYPE)*d0*d1*d2);
	return mat;
}

/**** set ****/

void set_1d(MAT*mat, UINT idx0, DTYPE val)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	cu_set << <1, 1 >> > (mat->data, idx0, val);
	hipDeviceSynchronize();
}
void set_2d(MAT*mat, UINT idx0, UINT idx1, DTYPE val)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	cu_set << <1, 1 >> > (mat->data, idx0 + (mat->d0)*idx1, val);
	hipDeviceSynchronize();
}
void set_3d(MAT*mat, UINT idx0, UINT idx1, UINT idx2, DTYPE val)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	cu_set << <1, 1 >> > (mat->data, idx0 + (mat->d0)*idx1 + (mat->d0)*(mat->d1)*idx2, val);
	hipDeviceSynchronize();
}

__global__ void cu_set(DTYPE*data, UINT idx, DTYPE val)
{
	data[idx] = val;
}

void cset_1d(CMAT*mat, UINT idx0, DTYPE re, DTYPE im)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	cu_cset << <1, 1 >> > (mat->data, idx0, re, im);
	hipDeviceSynchronize();
}
void cset_2d(CMAT*mat, UINT idx0, UINT idx1, DTYPE re, DTYPE im)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	cu_cset << <1, 1 >> > (mat->data, idx0 + (mat->d0)*idx1, re, im);
	hipDeviceSynchronize();
}
void cset_3d(CMAT*mat, UINT idx0, UINT idx1, UINT idx2, DTYPE re, DTYPE im)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	cu_cset << <1, 1 >> > (mat->data, idx0 + (mat->d0)*idx1 + (mat->d0)*(mat->d1)*idx2, re, im);
	hipDeviceSynchronize();
}

__global__ void cu_cset(CTYPE*data, UINT idx, DTYPE re, DTYPE im)
{
	data[idx].re = re;
	data[idx].im = im;
}


/**** fill ****/

void fill(MAT*mat, DTYPE val) // for real mat
{
#if DEBUG
	printf("%s\n", __func__);
	printf("max thread : %d\n", max_thread);
#endif
	UINT len = (mat->d0) * (mat->d1) * (mat->d2);
	UINT num_block = (UINT)(len / (UINT)max_thread) + 1;
	cu_fill << <num_block, max_thread >> > (mat->data, len - 1, val, max_thread);
	hipDeviceSynchronize();
}

__global__ void cu_fill(DTYPE* data, UINT len, DTYPE val, UINT size_block)
{
	ITER idx = threadIdx.x + blockIdx.x * size_block;
	if (idx > len)
		return;
	data[idx] = val;
}

void cfill(CMAT*mat, DTYPE re, DTYPE im) // for complex mat
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	UINT len = (mat->d0) * (mat->d1) * (mat->d2);

	UINT num_block = (len / max_thread) + 1;
	cu_cfill << <num_block, max_thread >> > (mat->data, len - 1, re, im, max_thread);
	hipDeviceSynchronize();
}
__global__ void cu_cfill(CTYPE* data, UINT len, DTYPE re, DTYPE im, UINT size_block)
{
	ITER idx = threadIdx.x + blockIdx.x * size_block;
	if (idx > len)
		return;
	data[idx].re = re;
	data[idx].im = im;
}

/**** get ****/

DTYPE get_1d(MAT*mat, UINT idx0)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	DTYPE ret;
	hipMemcpy(&ret, &(mat->data[idx0]), sizeof(DTYPE), hipMemcpyDeviceToHost);
	return ret;
}

DTYPE get_2d(MAT*mat, UINT idx0, UINT idx1)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	DTYPE ret;
	hipMemcpy(&ret, &(mat->data[idx0 + (mat->d0)*idx1]), sizeof(DTYPE), hipMemcpyDeviceToHost);
	return ret;
}
DTYPE get_3d(MAT*mat, UINT idx0, UINT idx1, UINT idx2)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	DTYPE ret;
	hipMemcpy(&ret, &(mat->data[idx0 + (mat->d0)*idx1 + (mat->d0)*(mat->d1)*idx2]), sizeof(DTYPE), hipMemcpyDeviceToHost);
	return ret;
}

CTYPE cget_1d(CMAT*mat, UINT idx0)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	CTYPE ret;
	hipMemcpy(&ret, &(mat->data[idx0]), sizeof(CTYPE), hipMemcpyDeviceToHost);
	return ret;
}

CTYPE cget_2d(MAT*mat, UINT idx0, UINT idx1)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	CTYPE ret;
	hipMemcpy(&ret, &(mat->data[idx0 + (mat->d0)*idx1]), sizeof(CTYPE), hipMemcpyDeviceToHost);
	return ret;
}
CTYPE cget_3d(MAT*mat, UINT idx0, UINT idx1, UINT idx2)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	CTYPE ret;
	hipMemcpy(&ret, &(mat->data[idx0 + (mat->d0)*idx1 + (mat->d0)*(mat->d1)*idx2]), sizeof(CTYPE), hipMemcpyDeviceToHost);
	return ret;
}
/**** submat ****/

void submat_1d(MAT* mat, MAT* submat, ITER d0_st, ITER d0_ed)
{
	UINT num_thread;
#if DEBUG
	printf("%s\n", __func__);
#endif
	if (mat->d0 < submat->d0) { printf("error in \nmat->d0 < submat->d0\n"); return; }
	if (d0_st == -1) d0_st = 0;
	if (d0_ed == -1) d0_ed = mat->d0;

	num_thread = (d0_ed - d0_st + 1) > max_thread ? max_thread : (d0_ed - d0_st + 1);
	cu_submat << <1, num_thread >> > (mat->data, submat->data, d0_st, d0_ed, 0, 0, mat->d0, 1, submat->d0, 1, num_thread);
	hipDeviceSynchronize();
}


void submat_2d(MAT* mat, MAT* submat,
	ITER d0_st, ITER d0_ed,
	ITER d1_st, ITER d1_ed)
{
	UINT num_thread;
#if DEBUG
	printf("%s\n", __func__);
#endif
	if (mat->d0 < submat->d0 || mat->d1 < submat->d1) { printf("error in \nmat->d0 < submat->d0 || mat->d1 < submat->d1 \n"); return; }
	if (d0_st == -1) d0_st = 0;
	if (d0_ed == -1) d0_ed = mat->d0;
	if (d1_st == -1) d1_st = 0;
	if (d1_ed == -1) d1_ed = mat->d1;

	num_thread = (d0_ed - d0_st + 1) > max_thread ? max_thread : (d0_ed - d0_st + 1);

	cu_submat << <d1_ed - d1_st + 1, num_thread >> > (mat->data, submat->data, d0_st, d0_ed, d1_st, 0, mat->d0, mat->d1, submat->d0, submat->d1, num_thread);
	hipDeviceSynchronize();
}


void submat_3d(MAT* mat, MAT* submat,
	ITER d0_st, ITER d0_ed,
	ITER d1_st, ITER d1_ed,
	ITER d2_st, ITER d2_ed)
{
	UINT num_thread;
	dim3 Dg;
#if DEBUG
	printf("%s\n", __func__);
#endif
	if (mat->d0 < submat->d0 || mat->d1 < submat->d1 || mat->d2 < submat->d2) { printf("error in \nmat->d0 < submat->d0 || mat->d1 < submat->d1 || mat->d2 < submat->d2\n"); return; }
	if (d0_st == -1) d0_st = 0;
	if (d0_ed == -1) d0_ed = mat->d0;
	if (d1_st == -1) d1_st = 0;
	if (d1_ed == -1) d1_ed = mat->d1;
	if (d2_st == -1) d2_st = 0;
	if (d2_ed == -1) d2_ed = mat->d2;

	Dg = dim3(d1_ed - d1_st + 1, d2_ed - d2_st + 1, 1);
	num_thread = (d0_ed - d0_st + 1) > max_thread ? max_thread : (d0_ed - d0_st + 1);
	cu_submat << <Dg, num_thread >> > (mat->data, submat->data, d0_st, d0_ed, d1_st, d2_st, mat->d0, mat->d1, submat->d0, submat->d1, num_thread);
	hipDeviceSynchronize();
}

__global__ void cu_submat(DTYPE*data, DTYPE*sub_data,
	ITER d0_st, ITER d0_ed,
	ITER d1_st, ITER d2_st,
	UINT mat_d0, UINT mat_d1,
	UINT sub_d0, UINT sub_d1,
	UINT num_thread
)
{
	ITER i, j, k;

	k = blockIdx.y;
	j = blockIdx.x;
	for (i = threadIdx.x; d0_st + i < d0_ed; i += num_thread)
	{
		sub_data[i + j * (sub_d0)+k * (sub_d0*sub_d1)]
			= data[(d0_st + i) + (d1_st + j)*(mat_d0)+(d2_st + k)*(mat_d0*mat_d1)];
	}
}

void csubmat_1d(CMAT* mat, CMAT* csubmat, ITER d0_st, ITER d0_ed)
{
	UINT num_thread;
#if DEBUG
	printf("%s\n", __func__);
#endif
	if (mat->d0 < csubmat->d0) { printf("error in \nmat->d0 < csubmat->d0\n"); return; }
	if (d0_st == -1) d0_st = 0;
	if (d0_ed == -1) d0_ed = mat->d0;

	num_thread = (d0_ed - d0_st + 1) > max_thread ? max_thread : (d0_ed - d0_st + 1);
	cu_csubmat << <1, num_thread >> > (mat->data, csubmat->data, d0_st, d0_ed, 0, 0, mat->d0, 1, csubmat->d0, 1, num_thread);
	hipDeviceSynchronize();
}


void csubmat_2d(CMAT* mat, CMAT* csubmat,
	ITER d0_st, ITER d0_ed,
	ITER d1_st, ITER d1_ed)
{
	UINT num_thread;
#if DEBUG
	printf("%s\n", __func__);
#endif
	if (mat->d0 < csubmat->d0 || mat->d1 < csubmat->d1) { printf("error in \nmat->d0 < csubmat->d0 || mat->d1 < csubmat->d1 \n"); return; }
	if (d0_st == -1) d0_st = 0;
	if (d0_ed == -1) d0_ed = mat->d0;
	if (d1_st == -1) d1_st = 0;
	if (d1_ed == -1) d1_ed = mat->d1;

	num_thread = (d0_ed - d0_st + 1) > max_thread ? max_thread : (d0_ed - d0_st + 1);

	cu_csubmat << <d1_ed - d1_st + 1, num_thread >> > (mat->data, csubmat->data, d0_st, d0_ed, d1_st, 0, mat->d0, mat->d1, csubmat->d0, csubmat->d1, num_thread);
	hipDeviceSynchronize();
}


void csubmat_3d(CMAT* mat, CMAT* csubmat,
	ITER d0_st, ITER d0_ed,
	ITER d1_st, ITER d1_ed,
	ITER d2_st, ITER d2_ed)
{
	UINT num_thread;
	dim3 Dg;
#if DEBUG
	printf("%s\n", __func__);
#endif
	if (mat->d0 < csubmat->d0 || mat->d1 < csubmat->d1 || mat->d2 < csubmat->d2) { printf("error in \nmat->d0 < csubmat->d0 || mat->d1 < csubmat->d1 || mat->d2 < csubmat->d2\n"); return; }
	if (d0_st == -1) d0_st = 0;
	if (d0_ed == -1) d0_ed = mat->d0;
	if (d1_st == -1) d1_st = 0;
	if (d1_ed == -1) d1_ed = mat->d1;
	if (d2_st == -1) d2_st = 0;
	if (d2_ed == -1) d2_ed = mat->d2;

	Dg = dim3(d1_ed - d1_st + 1, d2_ed - d2_st + 1, 1);
	num_thread = (d0_ed - d0_st + 1) > max_thread ? max_thread : (d0_ed - d0_st + 1);
	cu_csubmat << <Dg, num_thread >> > (mat->data, csubmat->data, d0_st, d0_ed, d1_st, d2_st, mat->d0, mat->d1, csubmat->d0, csubmat->d1, num_thread);
	hipDeviceSynchronize();
}

__global__ void cu_csubmat(CTYPE*data, CTYPE*sub_data,
	ITER d0_st, ITER d0_ed,
	ITER d1_st, ITER d2_st,
	UINT mat_d0, UINT mat_d1,
	UINT sub_d0, UINT sub_d1,
	UINT num_thread
)
{
	ITER i, j, k;

	k = blockIdx.y;
	j = blockIdx.x;
	for (i = threadIdx.x; d0_st + i < d0_ed; i += num_thread)
	{
		sub_data[i + j * (sub_d0)+k * (sub_d0*sub_d1)].re
			= data[(d0_st + i) + (d1_st + j)*(mat_d0)+(d2_st + k)*(mat_d0*mat_d1)].re;
		sub_data[i + j * (sub_d0)+k * (sub_d0*sub_d1)].im
			= data[(d0_st + i) + (d1_st + j)*(mat_d0)+(d2_st + k)*(mat_d0*mat_d1)].im;
	}
}


/**** miscellaneous  ****/
void free_MAT(MAT *mat)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	hipFree(mat->data);
	free(mat);
}

void free_CMAT(CMAT *mat)
{
#if DEBUG
	printf("%s\n", __func__);
#endif
	hipFree(mat->data);
	free(mat);
}


void print_MAT(MAT* mat)
{
#if DEBUG
	printf("%s\n", __func__);
#endif

	cu_print_MAT << <1, 1 >> > (mat->data, mat->d0, mat->d1, mat->d2);
	hipDeviceSynchronize();
}

__global__ void cu_print_MAT(DTYPE* data, UINT d0, UINT d1, UINT d2)
{
	ITER i, j, k;

	for (k = 0; k < d2; k++)
	{
		for (i = 0; i < d0; i++)
		{
			for (j = 0; j < d1; j++)
				printf("%.3lf ", data[k*(d1)*(d0)+j * (d0)+i]);
			printf("\n");
		}
		printf("\n");
	}
}

void print_CMAT(CMAT* mat)
{
#if DEBUG
	printf("%s\n", __func__);
#endif

	cu_print_CMAT << <1, 1 >> > (mat->data, mat->d0, mat->d1, mat->d2);
	hipDeviceSynchronize();
}

__global__ void cu_print_CMAT(CTYPE*data, UINT d0, UINT d1, UINT d2)
{
	ITER i, j, k;

	for (k = 0; k < d2; k++)
	{
		for (i = 0; i < d0; i++)
		{
			for (j = 0; j < d1; j++)
			{
				printf("%.3lf ", data[k*(d1)*(d0)+j * (d0)+i].re);
				printf("%.3lf|", data[k*(d1)*(d0)+j * (d0)+i].im);
			}printf("\n");
		}
		printf("\n");
	}
}


